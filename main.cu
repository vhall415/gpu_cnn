#include <stdio.h>
#include <stdint.h>
#include <hipDNN.h>
#include <opencv2/opencv.hpp>
#include <cassert>
#include <cstdlib>
#include <iostream>

// function to check for errors
#define checkCUDNN(expression) \
{                                \
    hipdnnStatus_t status = (expression);                        \
    if(status != HIPDNN_STATUS_SUCCESS) {                        \
        std::cerr << "Error on line " << __LINE__ < ": " << hipdnnGetErrorString(status) << std::endl;  \
        std::exit(EXIT_FAILURE);                                \
    }                                                           \
}

// use opencv to load/save an image from a path
cv::Mat load_image(const char* image_path) {
    cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_COLOR);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    return image;
}

void save_image(const char* output_filename, float* buffer, int height, int width) {
    cv::Mat output_image(height, width, CV_32FC3, buffer);
    //Make negative values zero
    cv::threshold(output_image, output_image, /*threshold=*/0, /*maxval=*/0, cv::THRESH_TOZERO);
    cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
    output_image.convertTo(output_image, CV_8UC3);
    std::cerr << "Wrote output to " << output_filename << std::endl;
}

int main(int argc, char* argv[]) {
    cv::Mat img = load_image("./tensor_flow.png");
    
    // create handle for cudnn
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // start of first convolutional layer -------------------------------------

    // create/set input tensor descriptor
    hipdnnTensorDescriptor_t in_desc;
    /*checkCUDNN(*/hipdnnCreateTensorDescriptor(&in_desc);
    /*checkCUDNN(*/hipdnnSetTensor4dDescriptor(in_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/1,
                                          /*channels=*/3,
                                          /*image_height=*/img.rows,
                                          /*image_width=*/img.cols);
    
    // create kernel descriptor
    hipdnnFilterDescriptor_t kernel_desc;
    /*checkCUDNN(*/hipdnnCreateFilterDescriptor(&kernel_desc);
    /*checkCUDNN(*/hipdnnSetFilter4dDescriptor(kernel_desc,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*format=*/HIPDNN_TENSOR_NCHW,
                                          /*out_channels=*/3,
                                          /*in_channels=*/3,
                                          /*kernel_height=*/3,  //5
                                          /*kernel_width=*/3); //5

    // create convolution descriptor
    hipdnnConvolutionDescriptor_t conv_desc;
    /*checkCUDNN(*/hipdnnCreateConvolutionDescriptor(&conv_desc);
    /*checkCUDNN(*/hipdnnSetConvolution2dDescriptor(conv_desc,
                                               /*pad_height=*/1,
                                               /*pad_width=*/1,
                                               /*vertical_stride=*/1,
                                               /*horizontal_stride=*/1,
                                               /*dilation_height=*/1,
                                               /*dilation_width=*/1,
                                               /*mode=*/HIPDNN_CONVOLUTION, //HIPDNN_CROSS_CORRELATION
                                               /*computeType=*/HIPDNN_DATA_FLOAT);

    // initialize variables for convolution
    int batch_size{0}, channels{0}, height{0}, width{0};
    /*checkCUDNN(*/hipdnnGetConvolution2dForwardOutputDim(conv_desc,
                                                     in_desc,
                                                     kernel_desc,
                                                     &batch_size,
                                                     &channels,
                                                     &height,
                                                     &width);

    std::cerr << "Output Image: " << height << " x " << width << " x " << channels << std::endl;

    // create output tensor descriptor
    hipdnnTensorDescriptor_t out_desc;
    /*checkCUDNN(*/hipdnnCreateTensorDescriptor(&out_desc);
    /*checkCUDNN(*/hipdnnSetTensor4dDescriptor(out_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/1,
                                          /*channels=*/3,
                                          /*image_height=*/img.rows,
                                          /*image_width=*/img.cols);

    // forward convolution algorithm ------------------------------------------
    hipdnnConvolutionFwdAlgo_t conv_alg;
    /*checkCUDNN(*/hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                   in_desc,
                                                   kernel_desc,
                                                   conv_desc,
                                                   out_desc,
                                                   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                   /*memoryLimitInBytes=*/0,
                                                   &conv_alg);
    
    // get forward convolution workspace size
    size_t workspace_bytes{0};
    /*checkCUDNN(*/hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       in_desc,
                                                       kernel_desc,
                                                       conv_desc,
						       out_desc,
                                                       conv_alg,
                                                       &workspace_bytes);
    std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB" << std::endl;
    assert(workspace_bytes > 0);

    // initialze device variables ---------------------------------------------
    void* d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    int image_bytes = batch_size * channels * height * width * sizeof(float);

    float* d_input{nullptr};
    hipMalloc(&d_input, image_bytes);
    hipMemcpy(d_input, img.ptr<float>(0), image_bytes, hipMemcpyHostToDevice);
    
    float* d_output{nullptr};
    hipMalloc(&d_output, image_bytes);
    hipMemset(d_output, 0, image_bytes);

    const float kernel_temp[3][3] = {
        {1, 1, 1},
        {1, -8, 1},
        {1, 1, 1}
    };

    float h_kernel[3][3][3][3];
    for(int kernel = 0; kernel < 3; ++kernel) {
        for(int channel = 0; channel < 3; ++channel) {
            for(int row = 0; row < 3; ++row) {
                for(int col = 0; col < 3; ++col) {
                    h_kernel[kernel][channel][row][col] = kernel_temp[row][col];
                }
            }
        }
    }

    float* d_kernel{nullptr};
    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

    const float alpha = 1.0f, beta = 0.0f;

    // forward convolution layer ----------------------------------------------

    /*checkCUDNN(*/hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       in_desc,
                                       d_input,
                                       kernel_desc,
                                       d_kernel,
                                       conv_desc,
                                       conv_alg,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       out_desc,
                                       d_output);

    // activation layer (RELU) -----------------------------------------------

    hipdnnActivationDescriptor_t act_desc;
    /*checkCUDNN(*/hipdnnCreateActivationDescriptor(&act_desc);
    /*checkCUDNN(*/hipdnnSetActivationDescriptor(act_desc,
                                            HIPDNN_ACTIVATION_SIGMOID,//HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN,
                                            /*relu_coef=*/0);

    /*checkCUDNN(*/hipdnnActivationForward(cudnn,
                                      act_desc,
                                      &alpha,
                                      out_desc,
                                      d_output,
                                      &beta,
                                      out_desc,
                                      d_output);



    float* h_output = new float[image_bytes];
    hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);

    save_image("cudnn_out.png", h_output, height, width);

    delete[] h_output;
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(in_desc);
    hipdnnDestroyTensorDescriptor(out_desc);
    hipdnnDestroyFilterDescriptor(kernel_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroyActivationDescriptor(act_desc);

    hipdnnDestroy(cudnn);
}
