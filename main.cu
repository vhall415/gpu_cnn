#include <stdio.h>
#include <stdint.h>
#include <hipDNN.h>
#include <opencv2/opencv.hpp>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <hipblas.h>

// function to check for errors
#define checkCUDNN(expression) 					\
{                                				\
    hipdnnStatus_t status = (expression);                        \
    if(status != HIPDNN_STATUS_SUCCESS) { 			\
	std::cerr << "Error on line " << __LINE__ << ": "	\
		  << hipdnnGetErrorString(status) << std::endl;	\
	std::exit(EXIT_FAILURE);				\
    }	\
}

// use opencv to load/save an image from a path
cv::Mat load_image(const char* image_path) {
    cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_UNCHANGED);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    std::cerr << "Input image: " << image.rows << " x " << image.cols << " x " << image.channels() << std::endl;
    return image;
}

void save_image(const char* output_filename, float* buffer, int height, int width) {
    cv::Mat output_image(height, width, CV_32FC3, buffer);
    //Make negative values zero
    cv::threshold(output_image, output_image, /*threshold=*/0, /*maxval=*/0, cv::THRESH_TOZERO);
    cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
    output_image.convertTo(output_image, CV_8UC3);
    cv::imwrite(output_filename, output_image);
    std::cerr << "Wrote output to " << output_filename << std::endl;
}

#define BATCH 1 // number of images
#define IN_CHANNELS 1   // number of channels of input image
#define FILTER_DIM 5    // side length of convolution filter size

int main(int argc, char* argv[]) {
    cv::Mat img = load_image("./gray_images/0.PNG");

    FILE *f;
    char buf[1000];
    // read weight files into arrays

    // conv1 weights
    // 5x5x1x32
    f = fopen("./weights/var0.txt", "r");
    float kernel_conv1[32][1][5][5];
    for(int kernel = 0; kernel < 32; kernel++) {
	for(int channel = 0; channel < 1; channel++) {
	    for(int row = 0; row < 5; row++) {
		for(int col = 0; col < 5; col++) {
		    if(fgets(buf,1000,f) != NULL)
		    	kernel_conv1[kernel][channel][row][col] = atof(buf);
		}
	    }
	}
    }
    fclose(f);
	
    // conv1 bias
    // 32
    float bias_conv1[32];
    f = fopen("./weights/var1.txt", "r");
    for(int i = 0; i < 32; i++) {
	if(fgets(buf, 1000, f) != NULL)
	    bias_conv1[i] = atof(buf);
    }
    fclose(f);

    // conv2 weigts
    // 5x5x32x64
    f = fopen("./weights/var2.txt", "r");
    float kernel_conv2[64][32][5][5];
    for(int kernel = 0; kernel < 64; kernel++) {
	for(int channel = 0; channel < 32; channel++) {
	    for(int row = 0; row < 5; row++) {
		for(int col = 0; col < 5; col++) {
		    if(fgets(buf, 1000, f) != NULL)
			kernel_conv2[kernel][channel][row][col] = atof(buf);
		}
	    }
	}
    }
    fclose(f);

    //conv2 bias
    // 64
    f = fopen("./weights/var3.txt", "r");
    float bias_conv2[64];
    for(int i = 0; i < 64; i++) {
	if(fgets(buf, 1000, f) != NULL)
	    bias_conv2[i] = atof(buf);
    }
    fclose(f);
    
    // fully connected layer weights
    // 3136x1024
    f = fopen("./weights/var4_0.txt", "r");
    float *fully_con = new float[3136*1024];
    for(int i = 0; i < 1568*1024; i++) {
	    if(fgets(buf,1000,f) != NULL)
	        fully_con[i] = atof(buf);
    }
    fclose(f);

    f = fopen("./weights/var4_1.txt", "r");
    for(int i = 1568*1024; i < 3136*1024; i++) {
	    if(fgets(buf,1000,f) != NULL)
	        fully_con[i] = atof(buf);
    }
    fclose(f);

    float (*fc_mat)[1024] = new float[3136][1024];

    for(int row = 0; row < 3136; row++) {
	    for(int col = 0; col < 1024; col++) {
	        fc_mat[row][col] = fully_con[1024*row+col];
	    }
    }

    delete[] fully_con;

    // fully connected layer bias
    // 1024
    f = fopen("./weights/var5.txt", "r");
    float bias_fc[1024];
    for(int i = 0; i < 1024; i++) {
	if(fgets(buf, 1000, f) != NULL)
	    bias_fc[i] = atof(buf);
    }
    fclose(f);

    // output layer weights
    // 1024x10
    f = fopen("./weights/var6.txt", "r");
    float out_mat[1024][10];
    for(int row = 0; row < 1024; row++) {
	for(int col = 0; col < 10; col++) {
	    if(fgets(buf,1000, f) != NULL)
	        out_mat[row][col] = atof(buf);
        }
    }
    fclose(f);

    // output layer bias
    // 10
    float bias_out[10];
    f = fopen("./weights/var7.txt", "r");
    for(int i = 0; i < 10; i++) {
        if(fgets(buf, 1000, f) != NULL) {
	        bias_out[i] = atof(buf);
	    }
    }
    fclose(f);

std::cerr << "1" << std::endl;

    // create handle for cudnn
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    // conv 1 descriptors -------------------------------------------------------------------

    // create/set input tensor descriptor
    hipdnnTensorDescriptor_t in_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&in_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(in_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/BATCH,
                                          /*channels=*/IN_CHANNELS,
                                          /*image_height=*/img.rows,
                                          /*image_width=*/img.cols));
    
    // create kernel descriptor
    hipdnnFilterDescriptor_t conv1_kernel_desc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&conv1_kernel_desc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(conv1_kernel_desc,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*format=*/HIPDNN_TENSOR_NCHW,
                                          /*out_channels=*/32,
                                          /*in_channels=*/IN_CHANNELS,
                                          /*kernel_height=*/FILTER_DIM,
                                          /*kernel_width=*/FILTER_DIM));

    // create convolution descriptor
    hipdnnConvolutionDescriptor_t conv1_desc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv1_desc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(conv1_desc,
                                               /*pad_height=*/2,
                                               /*pad_width=*/2,
                                               /*vertical_stride=*/1,
                                               /*horizontal_stride=*/1,
                                               /*dilation_height=*/1,
                                               /*dilation_width=*/1,
                                               /*mode=*/HIPDNN_CROSS_CORRELATION,
                                               /*computeType=*/HIPDNN_DATA_FLOAT));

    // initialize variables for convolution 1 output dimensions
    int conv1_batch{0}, conv1_chan{0}, conv1_h{0}, conv1_w{0};
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(conv1_desc,
                                                     in_desc,
                                                     conv1_kernel_desc,
                                                     &conv1_batch,
                                                     &conv1_chan,
                                                     &conv1_h,
                                                     &conv1_w));

    std::cerr << "Output Image: " << conv1_batch << " x " << conv1_h << " x " << conv1_w << " x " << conv1_chan << std::endl;

    // create output tensor descriptors
    hipdnnTensorDescriptor_t conv1_out_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&conv1_out_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(conv1_out_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/conv1_batch,
                                          /*channels=*/conv1_chan,
                                          /*image_height=*/conv1_h,
                                          /*image_width=*/conv1_w));

    hipdnnTensorDescriptor_t conv1_bias_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&conv1_bias_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(conv1_bias_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/conv1_batch,
                                          /*channels=*/conv1_chan,
                                          /*image_height=*/1,
                                          /*image_width=*/1));


    // get forward convolution algorithm
    hipdnnConvolutionFwdAlgo_t conv1_alg;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                   in_desc,
                                                   conv1_kernel_desc,
                                                   conv1_desc,
                                                   conv1_out_desc,
                                                   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                   /*memoryLimitInBytes=*/0,
                                                   &conv1_alg));
    
    // get forward convolution workspace size
    size_t conv1_work{0};
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       in_desc,
                                                       conv1_kernel_desc,
                                                       conv1_desc,
						                               conv1_out_desc,
                                                       conv1_alg,
                                                       &conv1_work));

    std::cerr << "Workspace size: " << (conv1_work / 1048576.0) << "MB" << std::endl;
    assert(conv1_work > 0);


    // relu descriptor -----------------------------------------------------------------------
    // same one used for all relu layers
    hipdnnActivationDescriptor_t act_desc;
    checkCUDNN(hipdnnCreateActivationDescriptor(&act_desc));
    checkCUDNN(hipdnnSetActivationDescriptor(act_desc,
				                            HIPDNN_ACTIVATION_RELU,
				                            HIPDNN_PROPAGATE_NAN,
				                            /*relu_coef=*/0));


    // pool1 descriptors ---------------------------------------------------------------------
    // get variables for pooling 1 output dimensions

    hipdnnPoolingDescriptor_t pool1_desc;
    checkCUDNN(hipdnnCreatePoolingDescriptor(&pool1_desc));
    checkCUDNN(hipdnnSetPooling2dDescriptor(pool1_desc,
				 /*mode=*/HIPDNN_POOLING_MAX,
				 /*maxpoolingNanOpt=*/HIPDNN_PROPAGATE_NAN,
				 /*windowHeight=*/2,
				 /*windowWidth=*/2,
				 /*verticalPadding=*/0,
				 /*horizontalPadding=*/0,
				 /*verticalStride=*/2,
				 /*horizontalStride=*/2));

    int pool1_batch{0}, pool1_chan{0}, pool1_h{0}, pool1_w{0};
    checkCUDNN(hipdnnGetPooling2dForwardOutputDim(pool1_desc,
                                      conv1_out_desc,
    				                  /*outN=*/&pool1_batch,
    			 	                  /*outC=*/&pool1_chan,
    				                  /*outH=*/&pool1_h,
    				                  /*outW=*/&pool1_w));

    std::cerr << "Pooling Output Size: " << pool1_batch << " x " << pool1_h << " x " << pool1_w << " x " << pool1_chan << std::endl;

    hipdnnTensorDescriptor_t pool1_out_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&pool1_out_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(pool1_out_desc,
			       /*format=*/HIPDNN_TENSOR_NHWC,
			       /*dataType=*/HIPDNN_DATA_FLOAT,
			       /*batch_size=*/pool1_batch,
			       /*channels=*/pool1_chan,
		   	       /*image_height=*/pool1_h,
			       /*image_width=*/pool1_w));
    
    // conv2 descriptors ---------------------------------------------------------------------
    
    hipdnnFilterDescriptor_t conv2_kernel_desc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&conv2_kernel_desc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(conv2_kernel_desc,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*format=*/HIPDNN_TENSOR_NCHW,
                                          /*out_channels=*/64,
                                          /*in_channels=*/pool1_chan,
                                          /*kernel_height=*/FILTER_DIM,
                                          /*kernel_width=*/FILTER_DIM));

    // create convolution descriptor
    hipdnnConvolutionDescriptor_t conv2_desc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv2_desc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(conv2_desc,
                                               /*pad_height=*/2,
                                               /*pad_width=*/2,
                                               /*vertical_stride=*/1,
                                               /*horizontal_stride=*/1,
                                               /*dilation_height=*/1,
                                               /*dilation_width=*/1,
                                               /*mode=*/HIPDNN_CROSS_CORRELATION,
                                               /*computeType=*/HIPDNN_DATA_FLOAT));

    // initialize variables for convolution 2 output dimensions
    int conv2_batch{0}, conv2_chan{0}, conv2_h{0}, conv2_w{0};
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(conv2_desc,
                                                     pool1_out_desc,
                                                     conv2_kernel_desc,
                                                     &conv2_batch,
                                                     &conv2_chan,
                                                     &conv2_h,
                                                     &conv2_w));

    std::cerr << "Conv2 Output Image: " << conv2_batch << " x " << conv2_h << " x " << conv2_w << " x " << conv2_chan << std::endl;

    // create output tensor descriptors
    hipdnnTensorDescriptor_t conv2_out_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&conv2_out_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(conv2_out_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/conv2_batch,
                                          /*channels=*/conv2_chan,
                                          /*image_height=*/conv2_h,
                                          /*image_width=*/conv2_w));

    hipdnnTensorDescriptor_t conv2_bias_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&conv2_bias_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(conv2_bias_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/conv2_batch,
                                          /*channels=*/conv2_chan,
                                          /*image_height=*/1,
                                          /*image_width=*/1));
    // get forward convolution algorithm
    hipdnnConvolutionFwdAlgo_t conv2_alg;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                   pool1_out_desc,
                                                   conv2_kernel_desc,
                                                   conv2_desc,
                                                   conv2_out_desc,
                                                   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                   /*memoryLimitInBytes=*/0,
                                                   &conv2_alg));
    
    // get forward convolution workspace size
    size_t conv2_work{0};
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       pool1_out_desc,
                                                       conv2_kernel_desc,
                                                       conv2_desc,
						                               conv2_out_desc,
                                                       conv2_alg,
                                                       &conv2_work));

    std::cerr << "Workspace size: " << (conv2_work / 1048576.0) << "MB" << std::endl;
    assert(conv2_work > 0);


    // pool2 descriptors ---------------------------------------------------------------------
    
    hipdnnPoolingDescriptor_t pool2_desc;
    checkCUDNN(hipdnnCreatePoolingDescriptor(&pool2_desc));
    checkCUDNN(hipdnnSetPooling2dDescriptor(pool2_desc,
				 /*mode=*/HIPDNN_POOLING_MAX,
				 /*maxpoolingNanOpt=*/HIPDNN_PROPAGATE_NAN,
				 /*windowHeight=*/2,
				 /*windowWidth=*/2,
				 /*verticalPadding=*/0,
				 /*horizontalPadding=*/0,
				 /*verticalStride=*/2,
				 /*horizontalStride=*/2));

    int pool2_batch{0}, pool2_chan{0}, pool2_h{0}, pool2_w{0};
    checkCUDNN(hipdnnGetPooling2dForwardOutputDim(pool2_desc,
                                      conv2_out_desc,
    				                  /*outN=*/&pool2_batch,
    			 	                  /*outC=*/&pool2_chan,
    				                  /*outH=*/&pool2_h,
    				                  /*outW=*/&pool2_w));

    std::cerr << "Pool2 Output Size: " << pool2_batch << " x " << pool2_h << " x " << pool2_w << " x " << pool2_chan << std::endl;

    hipdnnTensorDescriptor_t pool2_out_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&pool2_out_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(pool2_out_desc,
			       /*format=*/HIPDNN_TENSOR_NHWC,
			       /*dataType=*/HIPDNN_DATA_FLOAT,
			       /*batch_size=*/pool2_batch,
			       /*channels=*/pool2_chan,
		   	       /*image_height=*/pool2_h,
			       /*image_width=*/pool2_w));


    // fully connected layer descriptors -----------------------------------------------------
    
    hipblasStatus_t cublas_status;
    hipblasHandle_t cublas_handle;
    
    cublas_status = hipblasCreate(&cublas_handle);

    hipdnnTensorDescriptor_t fc_out_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&fc_out_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(fc_out_desc,
                               HIPDNN_TENSOR_NHWC,
                               HIPDNN_DATA_FLOAT,
                               pool2_batch,
                               1,
                               1,
                               1024));

    hipdnnTensorDescriptor_t fc_bias_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&fc_bias_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(fc_bias_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/pool2_batch,
                                          /*channels=*/1,
                                          /*image_height=*/1,
                                          /*image_width=*/1024));

    // dropout layer descriptors -------------------------------------------------------------

    //void* states;

    hipdnnDropoutDescriptor_t drop_desc;
    checkCUDNN(hipdnnCreateDropoutDescriptor(&drop_desc));
    checkCUDNN(hipdnnSetDropoutDescriptor(drop_desc,
                              cudnn,
                              /*dropout=*/0.1f,
                              /*states=*/NULL,
                              /*stateSizeInBytes=*/0,
                              /*seed=*/217));

    size_t drop_size{0};
    checkCUDNN(cudnnDropoutGetReserveSpaceSize(fc_out_desc,
                                    /*sizeInBytes=*/&drop_size));

    //size_t drop_state_size{0};
    //checkCUDNN(hipdnnDropoutGetStatesSize(cudnn,
    //                                    &drop_state_size));

    hipdnnTensorDescriptor_t drop_out_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&drop_out_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(drop_out_desc,
                               HIPDNN_TENSOR_NHWC,
                               HIPDNN_DATA_FLOAT,
                               pool2_batch,
                               1,
                               1,
                               1024));

    // output layer descriptors --------------------------------------------------------------

    hipdnnTensorDescriptor_t out_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&out_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(out_desc,
                               HIPDNN_TENSOR_NHWC,
                               HIPDNN_DATA_FLOAT,
                               pool2_batch,
                               1,
                               1,
                               10));

    hipdnnTensorDescriptor_t out_bias_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&out_bias_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(out_bias_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/pool2_batch,
                                          /*channels=*/1,
                                          /*image_height=*/1,
                                          /*image_width=*/10));
    // initialze device variables ---------------------------------------------
    int in_size = BATCH * IN_CHANNELS * img.rows * img.cols * sizeof(float);
    float* d_input{nullptr};
    hipMalloc(&d_input, in_size);
    hipMemcpy(d_input, img.ptr<float>(0), in_size, hipMemcpyHostToDevice);
    
    float* d_kernel_conv1{nullptr};
    hipMalloc(&d_kernel_conv1, sizeof(kernel_conv1));
    hipMemcpy(d_kernel_conv1, kernel_conv1, sizeof(kernel_conv1), hipMemcpyHostToDevice);
    
    void* d_conv1_work{nullptr};
    hipMalloc(&d_conv1_work, conv1_work);
    
    int conv1_size = conv1_batch * conv1_chan * conv1_h * conv1_w * sizeof(float);
    float* d_conv1_out{nullptr};
    hipMalloc(&d_conv1_out, conv1_size);
    hipMemset(d_conv1_out, 0, conv1_size);
    
    float* d_bias_conv1{nullptr};
    hipMalloc(&d_bias_conv1, sizeof(bias_conv1));
    hipMemcpy(d_bias_conv1, bias_conv1, sizeof(bias_conv1), hipMemcpyHostToDevice);
    
    int pool1_size = pool1_batch * pool1_chan * pool1_h * pool1_w * sizeof(float);
    float* d_pool1_out{nullptr};
    hipMalloc(&d_pool1_out, pool1_size);
    hipMemset(d_pool1_out, 0, pool1_size);

    float* d_kernel_conv2{nullptr};
    hipMalloc(&d_kernel_conv2, sizeof(kernel_conv2));
    hipMemcpy(d_kernel_conv2, kernel_conv2, sizeof(kernel_conv2), hipMemcpyHostToDevice);
    
    void* d_conv2_work{nullptr};
    hipMalloc(&d_conv2_work, conv2_work);
    
    int conv2_size = conv2_batch * conv2_chan * conv2_h * conv2_w * sizeof(float);
    float* d_conv2_out{nullptr};
    hipMalloc(&d_conv2_out, conv2_size);
    hipMemset(d_conv2_out, 0, conv2_size);

    float* d_bias_conv2{nullptr};
    hipMalloc(&d_bias_conv2, sizeof(bias_conv2));
    hipMemcpy(d_bias_conv2, bias_conv2, sizeof(bias_conv2), hipMemcpyHostToDevice);
    
    int pool2_size = pool2_batch * pool2_chan * pool2_h * pool2_w * sizeof(float);
    float* d_pool2_out{nullptr};
    hipMalloc(&d_pool2_out, pool2_size);
    hipMemset(d_pool2_out, 0, pool2_size);

    int fc_size = 3136*1024*sizeof(float);
    float* d_fully_con_mat{nullptr};
    hipMalloc(&d_fully_con_mat, fc_size);
    cublas_status = hipblasSetMatrix(3136, 1024, fc_size, fully_con, 3136, d_fully_con_mat, 3136);

    float* d_bias_fc{nullptr};
    hipMalloc(&d_bias_fc, sizeof(bias_fc));
    hipMemcpy(d_bias_fc, bias_fc, sizeof(bias_fc), hipMemcpyHostToDevice);
    
    int fc_out_size = 1024 * sizeof(float);
    float* d_fully_con_out{nullptr};
    hipMalloc(&d_fully_con_out, fc_out_size);
    hipMemset(d_fully_con_out, 0, fc_out_size);

    void* d_reserve{nullptr};
    hipMalloc(&d_reserve, drop_size);

    float* d_drop_out{nullptr};
    hipMalloc(&d_drop_out, fc_size);
    hipMemset(d_drop_out, 0, fc_size);

    int num_elems = pool2_chan * 1024 * 10;
    float* d_out_mat{nullptr};
    hipMalloc(&d_out_mat, num_elems);
    cublas_status = hipblasSetMatrix(1024, 10, num_elems*sizeof(float), out_mat, 1024, d_out_mat, 1024);

    float* d_bias_out{nullptr};
    hipMalloc(&d_bias_out, sizeof(bias_out));
    hipMemcpy(d_bias_out, bias_out, sizeof(bias_out), hipMemcpyHostToDevice);
    
    int out_size = 10 * sizeof(float);
    float* d_out{nullptr};
    hipMalloc(&d_out, out_size);
    hipMemset(d_out, 0, out_size);

    const float alpha = 1.0f, beta = 0.0f;

    // convolution 1 layer ----------------------------------------------
    // map grayscale input to 32 feature maps
    // 28x28x1 -> 28x28x32

    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       in_desc,
                                       d_input,
                                       conv1_kernel_desc,
                                       d_kernel_conv1,
                                       conv1_desc,
                                       conv1_alg,
                                       d_conv1_work,
                                       conv1_work,
                                       &beta,
                                       conv1_out_desc,
                                       d_conv1_out));

    checkCUDNN(hipdnnAddTensor(cudnn,
			      &alpha,
			      conv1_bias_desc,
   			      d_bias_conv1,
			      &alpha,
			      conv1_out_desc,
			      d_conv1_out));

    // relu 1 layer (activation) -----------------------------------------

    checkCUDNN(hipdnnActivationForward(cudnn,
			   act_desc,
			   &alpha,
			   conv1_out_desc,
			   d_conv1_out,
			   &beta,
			   conv1_out_desc,
			   d_conv1_out));

    // pooling 1 layer -------------------------------------------------
    // downsample by 2x
    // 28x28x32 -> 14x14x32
    
    checkCUDNN(hipdnnPoolingForward(cudnn,
			pool1_desc,
			&alpha,
			conv1_out_desc,
			d_conv1_out,
			&beta,
			pool1_out_desc,
			d_pool1_out));

    // convolution 2 layer -------------------------------------------
    // map 32 feature maps to 64
    // 2x2 padding
    // 14x14x32 -> 14x14x64

    checkCUDNN(hipdnnConvolutionForward(cudnn,
                            &alpha,
                            pool1_out_desc,
                            d_pool1_out,
                            conv2_kernel_desc,
                            d_kernel_conv2,
                            conv2_desc,
                            conv2_alg,
                            d_conv2_work,
                            conv2_work,
                            &beta,
                            conv2_out_desc,
                            d_conv2_out));

    checkCUDNN(hipdnnAddTensor(cudnn,
			      &alpha,
			      conv2_bias_desc,
   			      d_bias_conv2,
			      &alpha,
			      conv2_out_desc,
			      d_conv2_out));

    // relu 2 layer ---------------------------------------------------

    checkCUDNN(hipdnnActivationForward(cudnn,
                           act_desc,
                           &alpha,
                           conv2_out_desc,
                           d_conv2_out,
                           &beta,
                           conv2_out_desc,
                           d_conv2_out));

    // pooling 2 layer -----------------------------------------------
    // downsample by 2x
    // 14x14x64 -> 7x7x64

    checkCUDNN(hipdnnPoolingForward(cudnn,
                        pool2_desc,
                        &alpha,
                        conv2_out_desc,
                        d_conv2_out,
                        &beta,
                        pool2_out_desc,
                        d_pool2_out));

    // fully connected 1 layer ---------------------------------------
    // map 7x7x64 -> 1024 features

    cublas_status = hipblasSgemm(cublas_handle,
                                /*transa=*/HIPBLAS_OP_N,
                                /*transb=*/HIPBLAS_OP_N,
                                /*m=*/1,
                                /*n=*/7*7*64,
                                /*k=*/1024,
                                /*alpha=*/&alpha,
                                /*A=*/d_fully_con_mat,
                                /*lda=*/1,
                                /*B=*/d_pool2_out,
                                /*ldb=*/7*7*64,
                                /*beta=*/&beta,
                                /*C=*/d_fully_con_out,
                                /*ldc=*/1);

    checkCUDNN(hipdnnAddTensor(cudnn,
			      &alpha,
			      fc_bias_desc,
   			      d_bias_fc,
			      &alpha,
			      fc_out_desc,
			      d_fully_con_out));


    // relu 3 layer -------------------------------------------------


    checkCUDNN(hipdnnActivationForward(cudnn,
                           act_desc,
                           &alpha,
                           fc_out_desc,
                           d_fully_con_out,
                           &beta,
                           fc_out_desc,
                           d_fully_con_out));

    // dropout layer -----------------------------------------------


 
    // output layer --------------------------------------------------------------------------
    // map 1024 features to 10 classes (one for each digit)

    cublas_status = hipblasSgemm(cublas_handle,
                                /*transa=*/HIPBLAS_OP_N,
                                /*transb=*/HIPBLAS_OP_N,
                                /*m=*/1,
                                /*n=*/1024,
                                /*k=*/10,
                                /*alpha=*/&alpha,
                                /*A=*/d_fully_con_out,
                                /*lda=*/1,
                                /*B=*/d_out_mat,
                                /*ldb=*/1024,
                                /*beta=*/&beta,
                                /*C=*/d_out,
                                /*ldc=*/1);
    // add bias to d_out
    checkCUDNN(hipdnnAddTensor(cudnn,
			      &alpha,
			      out_bias_desc,
   			      d_bias_out,
			      &alpha,
			      out_desc,
			      d_out));


    //float* h_full_out = new float[fc_out_size];
    //hipMemcpy(h_full_out, d_fully_con_out, fc_out_size, hipMemcpyDeviceToHost);

    //std::cerr << h_full_out[0] << std::endl;

    float* h_out = new float[conv1_size];
    hipMemcpy(h_out, d_conv1_out, conv1_size, hipMemcpyDeviceToHost);
    
    save_image("./out.png", h_out, conv1_h, conv1_w);
    //delete[] h_full_out;
    delete[] h_out;
    delete[] fc_mat;

    hipFree(d_input);
    hipFree(d_kernel_conv1);
    hipFree(d_conv1_work);
    hipFree(d_conv1_out);
    hipFree(d_pool1_out);
    hipFree(d_kernel_conv2);
    hipFree(d_conv2_work);
    hipFree(d_conv2_out);
    hipFree(d_pool2_out);
    hipFree(d_fully_con_mat);
    hipFree(d_fully_con_out);
    hipFree(d_reserve);
    hipFree(d_drop_out);
    hipFree(d_out_mat);
    hipFree(d_out);
    hipFree(d_bias_conv1);
    hipFree(d_bias_conv2);
    hipFree(d_bias_fc);
    hipFree(d_bias_out);

    hipblasDestroy(cublas_handle);

    hipdnnDestroyTensorDescriptor(in_desc);
    hipdnnDestroyFilterDescriptor(conv1_kernel_desc);
    hipdnnDestroyConvolutionDescriptor(conv1_desc);
    hipdnnDestroyTensorDescriptor(conv1_out_desc);
    hipdnnDestroyActivationDescriptor(act_desc);
    hipdnnDestroyPoolingDescriptor(pool1_desc);
    hipdnnDestroyTensorDescriptor(pool1_out_desc);
    hipdnnDestroyFilterDescriptor(conv2_kernel_desc);
    hipdnnDestroyConvolutionDescriptor(conv2_desc);
    hipdnnDestroyTensorDescriptor(conv2_out_desc);
    hipdnnDestroyPoolingDescriptor(pool2_desc);
    hipdnnDestroyTensorDescriptor(pool2_out_desc);
    hipdnnDestroyTensorDescriptor(fc_out_desc);
    hipdnnDestroyDropoutDescriptor(drop_desc);
    hipdnnDestroyTensorDescriptor(drop_out_desc);
    hipdnnDestroyTensorDescriptor(out_desc);
    hipdnnDestroyTensorDescriptor(conv1_bias_desc);
    hipdnnDestroyTensorDescriptor(conv2_bias_desc);
    hipdnnDestroyTensorDescriptor(fc_bias_desc);
    hipdnnDestroyTensorDescriptor(out_bias_desc);

    
    hipdnnDestroy(cudnn);
}
