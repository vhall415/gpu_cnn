#include <stdio.h>
#include <stdint.h>
#include <hipDNN.h>
#include <opencv2/opencv.hpp>
#include <cassert>
#include <cstdlib>
#include <iostream>

// function to check for errors
#define checkCUDNN(expression) \
{                                \
    hipdnnStatus_t status = (expression);                        \
    if(status != HIPDNN_STATUS_SUCCESS) {                        \
        std::cerr << "Error on line " << __LINE__ < ": " << hipdnnGetErrorString(status) << std::endl;  \
        std::exit(EXIT_FAILURE);                                \
    }                                                           \
}

// use opencv to load/save an image from a path
cv::Mat load_image(const char* image_path) {
    cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_UNCHANGED);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    std::cerr << "Input image: " << image.rows << " x " << image.cols << " x " << image.channels() << std::endl;
    return image;
}

void save_image(const char* output_filename, float* buffer, int height, int width) {
    cv::Mat output_image(height, width, CV_32FC3, buffer);
    //Make negative values zero
    cv::threshold(output_image, output_image, /*threshold=*/0, /*maxval=*/0, cv::THRESH_TOZERO);
    cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
    output_image.convertTo(output_image, CV_8UC3);
    cv::imwrite(output_filename, output_image);
    std::cerr << "Wrote output to " << output_filename << std::endl;
}

int main(int argc, char* argv[]) {
    cv::Mat img = load_image("./bw_images/0.PNG");

    FILE *f;
    char buf[1000];
    // read weight files into arrays
    // 5x5x1x32
    f = fopen("./weights/var0.txt", "r");
    float kernel_conv1[32][1][5][5];
    for(int kernel = 0; kernel < 32; kernel++) {
	for(int channel = 0; channel < 1; channel++) {
	    for(int row = 0; row < 5; row++) {
		for(int col = 0; col < 5; col++) {
		    if(fgets(buf,1000,f) != NULL)
		    	kernel_conv1[kernel][channel][row][col] = atof(buf);
		}
	    }
	}
    }
    fclose(f);
	
    // 32
    float bias_conv1[32];
    f = fopen("./weights/var1.txt", "r");
    for(int i = 0; i < 32; i++) {
	if(fgets(buf, 1000, f) != NULL)
	    bias_conv1[i] = atof(buf);
    }
    fclose(f);

    // 5x5x32x64
    f = fopen("./weights/var2.txt", "r");
    float kernel_conv2[64][32][5][5];
    for(int kernel = 0; kernel < 64; kernel++) {
	for(int channel = 0; channel < 32; channel++) {
	    for(int row = 0; row < 5; row++) {
		for(int col = 0; col < 5; col++) {
		    if(fgets(buf, 1000, f) != NULL)
			kernel_conv2[kernel][channel][row][col] = atof(buf);
		}
	    }
	}
    }
    fclose(f);

    // 64
    f = fopen("./weights/var3.txt", "r");
    float bias_conv2[64];
    for(int i = 0; i < 64; i++) {
	if(fgets(buf, 1000, f) != NULL)
	    bias_conv2[i] = atof(buf);
    }
    fclose(f);

    // 3136x1024
    f = fopen("./weights/var4.txt", "r");
    float fully_con[3136][1024];
    for(int row = 0; row < 3136; row++) {
	for(int col = 0; col < 1024; col++) {
 	    if(fgets(buf, 1000, f) != NULL)
	        fully_con[row][col] = atof(buf);
	}
    }
    fclose(f);

    // 1024
    f = fopen("./weights/var5.txt", "r");
    float bias_fully_con[1024];
    for(int i = 0; i < 1024; i++) {
	if(fgets(buf, 1000, f) != NULL)
	    bias_fully_con[i] = atof(buf);
    }
    fclose(f);

    // 1024x10
    f = fopen("./weights/var6.txt", "r");
    float drop[1024][10];
    for(int row = 0; row < 1024; row++) {
	for(int col = 0; col < 10; col++) {
	    if(fgets(buf,1000, f) != NULL)
	        drop[row][col] = atof(buf);
        }
    }
    fclose(f);

    // 10
    float softmax[10];
    f = fopen("./weights/var7.txt", "r");
    for(int i = 0; i < 10; i++) {
        if(fgets(buf, 1000, f) != NULL) {
	    softmax[i] = atof(buf);
	}
    }
    fclose(f);

    // create handle for cudnn
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // start of first convolutional layer -------------------------------------

    // create/set input tensor descriptor
    hipdnnTensorDescriptor_t in_desc;
    /*checkCUDNN(*/hipdnnCreateTensorDescriptor(&in_desc);
    /*checkCUDNN(*/hipdnnSetTensor4dDescriptor(in_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/1,
                                          /*channels=*/1,
                                          /*image_height=*/img.rows,
                                          /*image_width=*/img.cols);
    
    // create kernel descriptor
    hipdnnFilterDescriptor_t conv1_kernel_desc;
    /*checkCUDNN(*/hipdnnCreateFilterDescriptor(&conv1_kernel_desc);
    /*checkCUDNN(*/hipdnnSetFilter4dDescriptor(conv1_kernel_desc,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*format=*/HIPDNN_TENSOR_NCHW,
                                          /*out_channels=*/32,
                                          /*in_channels=*/1,
                                          /*kernel_height=*/5,
                                          /*kernel_width=*/5);

    // create convolution descriptor
    hipdnnConvolutionDescriptor_t conv1_desc;
    /*checkCUDNN(*/hipdnnCreateConvolutionDescriptor(&conv1_desc);
    /*checkCUDNN(*/hipdnnSetConvolution2dDescriptor(conv1_desc,
                                               /*pad_height=*/2,
                                               /*pad_width=*/2,
                                               /*vertical_stride=*/1,
                                               /*horizontal_stride=*/1,
                                               /*dilation_height=*/1,
                                               /*dilation_width=*/1,
                                               /*mode=*/HIPDNN_CROSS_CORRELATION,
                                               /*computeType=*/HIPDNN_DATA_FLOAT);

    // initialize variables for convolution 1 output dimensions
    int batch_size{0}, channels{0}, height{0}, width{0};
    /*checkCUDNN(*/hipdnnGetConvolution2dForwardOutputDim(conv1_desc,
                                                     in_desc,
                                                     conv1_kernel_desc,
                                                     &batch_size,
                                                     &channels,
                                                     &height,
                                                     &width);

    std::cerr << "Output Image: " << conv1_batch << " x " << conv1_h << " x " << width << " x " << channels << std::endl;

    // create output tensor descriptors
    hipdnnTensorDescriptor_t conv1_out_desc;
    /*checkCUDNN(*/hipdnnCreateTensorDescriptor(&conv1_out_desc);
    /*checkCUDNN(*/hipdnnSetTensor4dDescriptor(conv1_out_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/batch_size,
                                          /*channels=*/channels,
                                          /*image_height=*/height,
                                          /*image_width=*/width);

    // get variables for pooling 1 output dimensions
 
    hipdnnPoolingDescriptor_t pool1_desc;
    hipdnnCreatePoolingDescriptor(&pool1_desc);
    hipdnnSetActivationDescriptor(pool1_desc,
				 /*mode=*/HIPDNN_POOLING_MAX,
				 /*maxpoolingNanOpt=*/HIPDNN_PROPAGATE_NAN,
				 /*windowHeight=*/2,
				 /*windowWidth=*/2,
				 /*verticalPadding=*/1,
				 /*horizontalPadding=*/1,
				 /*verticalStride=*/2,
				 /*horizontalStride=*/2);

    int pool1_batch{0}, pool1_chan{0}, pool1_h{0}, pool1_w{0};
    hipdnnGetPooling2dForwardOutputDim(pool_desc,
                                      conv1_out_desc,
    				      /*outN=*/&pool1_batch,
    			 	      /*outC=*/&pool1_chan,
    				      /*outH=*/&pool1_h,
    				      /*outW=*/&pool1_w)

    std::cerr << "Pooling Output Size: " << pool1_batch << " x " << pool1_h << " x " << pool1_w << " x " << pool1_chan << std::endl;

    hipdnnTensorDescriptor_t pool1_out_desc;
    hipdnnCreateTensorDescriptor(&pool1_out_desc);
    hipdnnSetTensor4dDescriptor(pool1_out_desc,
			       /*format=*/HIPDNN_TENSOR_NHWC,
			       /*dataType=*/HIPDNN_DATA_FLOAT,
			       /*batch_size=*/pool1_batch,
			       /*channels=*/pool1_chan,
		   	       /*image_height=*/pool1_h,
			       /*image_width=*/pool1_w);
    
    hipdnnTensorDescriptor_t conv2_out_desc;
    /*checkCUDNN(*/hipdnnCreateTensorDescriptor(&conv2_out_desc);
    /*checkCUDNN(*/hipdnnSetTensor4dDescriptor(conv2_out_desc,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/1,
                                          /*channels=*/64,
                                          /*image_height=*/img.rows/2,
                                          /*image_width=*/img.cols/2);

    hipdnnTensorDescriptor_t pool2_out_desc;
    hipdnnCreateTensorDescriptor(&pool2_out_desc);
    hipdnnSetTensor4dDescriptor(pool2_out_desc,
			       /*format=*/HIPDNN_TENSOR_NHWC,
			       /*dataType=*/HIPDNN_DATA_FLOAT,
			       /*batch_size=*/1,
			       /*channels=*/64,
		   	       /*image_height=*/img.rows/4,
			       /*image_width=*/img.cols/4);
 

    // get forward convolution algorithm
    hipdnnConvolutionFwdAlgo_t conv_alg;
    /*checkCUDNN(*/hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                   in_desc,
                                                   kernel_desc,
                                                   conv_desc,
                                                   conv1_out_desc,
                                                   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                   /*memoryLimitInBytes=*/0,
                                                   &conv_alg);
    
    // get forward convolution workspace size
    size_t workspace_bytes{0};
    /*checkCUDNN(*/hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       in_desc,
                                                       kernel_desc,
                                                       conv_desc,
						       conv1_out_desc,
                                                       conv_alg,
                                                       &workspace_bytes);
    std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB" << std::endl;
    assert(workspace_bytes > 0);

    // initialze device variables ---------------------------------------------
    void* d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    int image_bytes = batch_size * channels * height * width * sizeof(float);
    std::cout << "image_bytes: " << image_bytes << std::endl;

    float* d_input{nullptr};
    hipMalloc(&d_input, image_bytes);
    hipMemcpy(d_input, img.ptr<float>(0), image_bytes, hipMemcpyHostToDevice);
    
    float* d_conv1_out{nullptr};
    hipMalloc(&d_conv1_out, image_bytes);
    hipMemset(d_conv1_out, 0, image_bytes);

    float* d_kernel_conv1{nullptr};
    hipMalloc(&d_kernel_conv1, sizeof(kernel_conv1));
    hipMemcpy(d_kernel_conv1, kernel_conv1, sizeof(kernel_conv1), hipMemcpyHostToDevice);

    float* d_kernel_conv2{nullptr};
    hipMalloc(&d_kernel_conv2, sizeof(kernel_conv2));
    hipMemcpy(d_kernel_conv2, kernel_conv2, sizeof(kernel_conv2), hipMemcpyHostToDevice);
    
    int pool1_size = pool1_batch * pool1_chan * pool1_h * pool1_w * sizeof(float);
    float* d_pool1_out{nullptr};
    hipMalloc(&d_pool1_out, pool1_size);
    hipMemset(d_pool1_out, 0, pool1_size);


    const float alpha = 1.0f, beta = 0.0f;

    // convolution 1 layer ----------------------------------------------
    // map grayscale input to 32 feature maps
    // 28x28x1 -> 28x28x32

    /*checkCUDNN(*/hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       in_desc,
                                       d_input,
                                       kernel_desc,
                                       d_kernel_conv1,
                                       conv_desc,
                                       conv_alg,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       conv1_out_desc,
                                       d_conv1_out);

    // relu 1 layer (activation) -----------------------------------------

    hipdnnActivationDescriptor_t act_desc;
    hipdnnCreateActivationDescriptor(&act_desc);
    hipdnnSetActivationDescriptor(act_desc,
				 HIPDNN_ACTIVATION_RELU,
				 HIPDNN_PROPAGATE_NAN,
				 /*relu_coef=*/0);

    hipdnnActivationForward(cudnn,
			   act_desc,
			   &alpha,
			   conv1_out_desc,
			   &d_conv1_out,
			   &beta,
			   conv1_out_desc,
			   d_conv1_out);

    // pooling 1 layer -------------------------------------------------
    // downsample by 2x
    // 28x28x32 -> 14x14x32
    

    hipdnnPoolingForward(cudnn,
			pool_desc,
			&alpha,
			conv1_out_desc,
			d_conv1_out,
			&beta,
			pool1_out_desc,
			d_pool1_out);

    // convolution 2 layer -------------------------------------------
    // map 32 feature maps to 64
    // 2x2 padding
    // 14x14x32 -> 14x14x64



    // relu 2 layer ---------------------------------------------------



    // pooling 2 layer -----------------------------------------------
    // downsample by 2x
    // 14x14x64 -> 7x7x64



    // fully connected 1 layer ---------------------------------------
    // map 7x7x64 -> 1024 features



    // relu 3 layer -------------------------------------------------



    // dropout layer -----------------------------------------------
    // control complexity of model



    // softmax layer -----------------------------------------------
    // map 1024 features to 10 classes (one for each digit)



    float* h_output = new float[image_bytes];
    hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);
   
    save_image("./conv1_out.png", h_output, height, width);

    delete[] h_output;
    hipFree(d_kernel_conv1);
    hipFree(d_kernel_conv2);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(in_desc);
    hipdnnDestroyTensorDescriptor(out_desc);
    hipdnnDestroyFilterDescriptor(kernel_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);

    hipdnnDestroy(cudnn);
}
